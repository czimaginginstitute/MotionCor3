#include "hip/hip_runtime.h"
#include "CCorrectInc.h"
#include <memory.h>
#include <stdio.h>
#include <pthread.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <Util/Util_Time.h>
#include <nvToolsExt.h>

using namespace MotionCor2;
using namespace MotionCor2::Correct;

// padded sizeX, sizeY, number of frames, number of patches
static __device__ __constant__ int giSizes[4];

static __global__ void mGCorrect3D
(	float fBFactor,
	float* gfPadFrmIn,
	float* gfPatCenters,
	float* gfPatShifts,
	bool* gbBadShifts,
	int iUpSample,
	float* gfPadFrmOut
)
{	int y = blockIdx.y * blockDim.y + threadIdx.y;
        if(y >= giSizes[1]) return;
	int iOut = y * giSizes[0] + blockIdx.x;
	//-------------------------------------
	float afXYZ[2] = {0.0f};
	float fSx = 0.0f, fSy = 0.0f, fW = 0.0f;
	for(int p=0; p<giSizes[3]; p++)
	{	if(gbBadShifts != 0L && gbBadShifts[p]) continue;
		//-----------------------------------------------
		int k =  p * 2;
		afXYZ[0] = (blockIdx.x - gfPatCenters[k]) / gridDim.x;
		afXYZ[1] = (y - gfPatCenters[k+1]) / giSizes[1];
		afXYZ[0] = sqrtf(afXYZ[0] * afXYZ[0] + afXYZ[1] * afXYZ[1]);
		if(afXYZ[0] > 0.6f) continue;
		//----------------------------
		afXYZ[0] = expf(-fBFactor * afXYZ[0]);
		fW += afXYZ[0];
		//-------------
		fSx += gfPatShifts[p * 2] * afXYZ[0];
		fSy += gfPatShifts[p * 2 + 1] * afXYZ[0];
	}
	if(fW > 0)
	{	fSx = fSx / fW;
		fSy = fSy / fW;
	}
	//----------------------
	int x = (int)((blockIdx.x - fSx) * iUpSample);
	y = (int)((y - fSy) * iUpSample);
	int iSizeX = gridDim.x * iUpSample;
	int iSizeY = giSizes[1] * iUpSample;
	//----------------------
	if(x < 0 || y < 0 || x >= iSizeX || y >= iSizeY)
	{	x = (x < 0) ? -x : x;
		y = (y < 0) ? -y : y;
		x = (811 * x) % iSizeX;
		y = (811 * y) % iSizeY;
	}
	//----------------------
	iSizeX = (iSizeX / 2 + 1) * 2;
	gfPadFrmOut[iOut] = gfPadFrmIn[y * iSizeX + x];
}

Align::CPatchShifts* GCorrectPatchShift::m_pPatchShifts = 0L;

GCorrectPatchShift::GCorrectPatchShift(void)
{
	m_aBlockDim.x = 1;
	m_aBlockDim.y = 64;
	m_iUpsample = 1;
	//---------------------------
	m_pUpsInvFFT = 0L;
	m_gCmpUpsampled = 0L;
}

GCorrectPatchShift::~GCorrectPatchShift(void)
{
	mClean();
}

void GCorrectPatchShift::mClean(void)
{
        if(m_gCmpUpsampled != 0L)
        {       hipFree(m_gCmpUpsampled);
                m_gCmpUpsampled = 0L;
        }
	if(m_pUpsInvFFT != 0L)
	{	delete m_pUpsInvFFT;
		m_pUpsInvFFT = 0L;
	}
}

void GCorrectPatchShift::DoIt
(	Align::CPatchShifts* pPatchShifts,
	DU::CDataPackage* pPackage
)
{	nvtxRangePushA("GCorrectPatchShift::DoIt");
	Util_Time utilTime; utilTime.Measure();
	//-------------------------------------
	m_pPatchShifts = pPatchShifts;
	m_pPackage = pPackage;
	m_pFullShift = pPatchShifts->m_pFullShift;
	m_aiOutCmpSize[0] = m_pPackage->m_pAlnSums->m_aiStkSize[0] / 2 + 1;
	m_aiOutCmpSize[1] = m_pPackage->m_pAlnSums->m_aiStkSize[1];
	m_aiOutPadSize[0] = m_aiOutCmpSize[0] * 2;
	m_aiOutPadSize[1] = m_aiOutCmpSize[1];
	//------------------------------------
	CBufferPool* pBufferPool = CBufferPool::GetInstance();
	CStackBuffer* pFrmBuffer = pBufferPool->GetBuffer(EBuffer::frm);
	m_aiInCmpSize[0] = pFrmBuffer->m_aiCmpSize[0];
	m_aiInCmpSize[1] = pFrmBuffer->m_aiCmpSize[1];
	m_aiInPadSize[0] = m_aiInCmpSize[0] * 2;
	m_aiInPadSize[1] = m_aiInCmpSize[1];
	//----------------------------------
	int iNumGpus = pBufferPool->m_iNumGpus;
	GCorrectPatchShift* pThreads = new GCorrectPatchShift[iNumGpus];
	for(int i=0; i<iNumGpus; i++)
	{	pThreads[i].Run(i);
	}
	for(int i=0; i<iNumGpus; i++)
	{	pThreads[i].WaitForExit(-1.0f);
	}
	delete[] pThreads;
	//-----------------
	mSumPartialSums();
	mCorrectMag();
	mUnpadSums();
	//-----------------
	float fSecs = utilTime.GetElapsedSeconds();
	printf("Correction of local motion: %f sec\n\n", fSecs); 
	nvtxRangePop();
}

void GCorrectPatchShift::Run(int iNthGpu)
{
	m_iNthGpu = iNthGpu;
	this->Start();
}

void GCorrectPatchShift::ThreadMain(void)
{
	CCorrectFullShift::mInit();
	mSetupUpSample();
	//-------------------------
	CBufferPool* pBufferPool = CBufferPool::GetInstance();
	m_pForwardFFT = pBufferPool->GetForwardFFT(m_iNthGpu);
	m_pForwardFFT->CreateForwardPlan(m_aiInPadSize, true);
	//----------------------------------------------------
	int aiSizes[] = 
	{ m_aiInPadSize[0], m_aiInPadSize[1], 
	  m_pPatchShifts->m_aiFullSize[2],
	  m_pPatchShifts->m_iNumPatches
	};
	hipMemcpyToSymbol(HIP_SYMBOL(giSizes), aiSizes, sizeof(aiSizes));
	//----------------------------------------------------
	int iNumPoints = m_pPatchShifts->m_iNumPatches *
	   m_pPatchShifts->m_aiFullSize[2];
	int iBytes = iNumPoints * (2 * sizeof(float) + sizeof(bool));
	hipMalloc(&m_gfPatShifts, iBytes);
	m_gbBadShifts = (bool*)(m_gfPatShifts + iNumPoints * 2);
	//------------------------------------------------------
	m_pPatchShifts->CopyShiftsToGpu(m_gfPatShifts);
	m_pPatchShifts->CopyFlagsToGpu(m_gbBadShifts);
	//------------------------------------------------------------
	iBytes = m_pPatchShifts->m_iNumPatches * 2 * sizeof(float);
	hipMalloc(&m_gfPatCenters, iBytes);
	//----------------------------------
	m_pPatchShifts->CopyCentersToGpu(m_gfPatCenters);
	//-----------------------------------------------
	m_aGridDim.x = (m_aiInCmpSize[0] - 1) * 2;
	m_aGridDim.y = (m_aiInCmpSize[1] + m_aBlockDim.y - 1) / m_aBlockDim.y;
	//--------------------------------------------------------------------
	mCorrectCpuFrames();
	mCorrectGpuFrames();
	//------------------
	CCorrectFullShift::Wait();
	if(m_gfPatShifts != 0L) hipFree(m_gfPatShifts);
	if(m_gfPatCenters != 0L) hipFree(m_gfPatCenters);
	mClean();
}

void GCorrectPatchShift::mCorrectCpuFrames(void)
{
	int iCount = 0;
	int iStartFrm = m_pFrmBuffer->GetStartFrame(m_iNthGpu);
	int iNumFrames = m_pFrmBuffer->GetNumFrames(m_iNthGpu);
	size_t tBytes = m_pFrmBuffer->m_tFmBytes;
	hipfftComplex* pCmpFrm = 0L;
	hipfftComplex* gCmpBuf = m_pTmpBuffer->GetFrame(m_iNthGpu, 0); 
	hipfftComplex* gCmpAln = m_pTmpBuffer->GetFrame(m_iNthGpu, 1);
	//-----------------------------------------------------------
	for(int i=0; i<iNumFrames; i++)
	{	if(m_pFrmBuffer->IsGpuFrame(m_iNthGpu, i)) continue;
		pCmpFrm = m_pFrmBuffer->GetFrame(m_iNthGpu, i);
		//---------------------------------------------
		m_iAbsFrm = iStartFrm + i;
		int iStream = iCount % 2;
		//-----------------------
		if(iStream == 1) hipStreamSynchronize(m_aStreams[0]);
		hipMemcpyAsync(gCmpBuf, pCmpFrm, tBytes, 
		   hipMemcpyDefault, m_aStreams[iStream]);
		if(iStream == 1) hipStreamSynchronize(m_aStreams[1]);
		//----------------------------------------------------
		mAlignFrame(gCmpBuf);
		mGenSums(gCmpAln);
		iCount += 1;	
	}
}

void GCorrectPatchShift::mCorrectGpuFrames(void)
{
	int iStartFrm = m_pFrmBuffer->GetStartFrame(m_iNthGpu);
	int iNumFrames = m_pFrmBuffer->GetNumFrames(m_iNthGpu);
	hipfftComplex* gCmpAln = m_pTmpBuffer->GetFrame(m_iNthGpu, 1);
	for(int i=0; i<iNumFrames; i++)
	{	if(!m_pFrmBuffer->IsGpuFrame(m_iNthGpu, i)) continue;
		//---------------------------------------------------
		m_iAbsFrm = iStartFrm + i;
		hipfftComplex* gCmpFrm = m_pFrmBuffer->GetFrame(m_iNthGpu, i);
		mAlignFrame(gCmpFrm);
		mGenSums(gCmpAln);	
	}
}

//--------------------------------------------------------------------
// Upsample the input frame if the motioncor binning is 1 and the
// frame size does not exceed 8K x 8k.
//--------------------------------------------------------------------
void GCorrectPatchShift::mSetupUpSample(void)
{
	m_iUpsample = 2;
	CInput* pInput = CInput::GetInstance();
	if(pInput->m_fFourierBin >= 1.5f) m_iUpsample = 1;
	//---------------------------
	int iImgSize = (m_aiInCmpSize[0] - 1) * 2;
	if(iImgSize > m_aiInCmpSize[1]) iImgSize = m_aiInCmpSize[1];
	if(iImgSize >= 8192) m_iUpsample = 1;
	//---------------------------
	m_iUpsample = 1; //////////////////////////////////////////
	m_aiUpCmpSize[0] = (m_aiInCmpSize[0] - 1) * m_iUpsample + 1;
	m_aiUpCmpSize[1] = m_aiInCmpSize[1] * m_iUpsample;
	//---------------------------
	size_t tBytes = sizeof(hipfftComplex) * m_aiUpCmpSize[0]
	   * m_aiUpCmpSize[1];
	hipMalloc(&m_gCmpUpsampled, tBytes);
	//---------------------------
	m_pUpsInvFFT = new Util::CCufft2D;
	m_pUpsInvFFT->CreateInversePlan(m_aiUpCmpSize, true);
}

void GCorrectPatchShift::mUpSample(hipfftComplex* gCmpFrm)
{
	Util::GFtResize2D ftResize2D;
	ftResize2D.UpSample(gCmpFrm, m_aiInCmpSize,
	   m_gCmpUpsampled, m_aiUpCmpSize, m_aStreams[0]);
	//---------------------------
	m_pUpsInvFFT->Inverse(m_gCmpUpsampled, m_aStreams[0]);
}

void GCorrectPatchShift::mAlignFrame(hipfftComplex* gCmpFrm)
{
	float fBFactor = 100.0f;
	//---------------------------
	mUpSample(gCmpFrm);
	float* gfUpsampled = reinterpret_cast<float*>(m_gCmpUpsampled);
	//float* gfPadFrm = reinterpret_cast<float*>(gCmpFrm);
	//---------------------------
	CBufferPool* pBufferPool = CBufferPool::GetInstance();
	float* gfPadAln = (float*)m_pTmpBuffer->GetFrame(m_iNthGpu, 1);
	//---------------------------
	int iOffset = m_iAbsFrm * m_pPatchShifts->m_iNumPatches;
	float* gfPatShifts = m_gfPatShifts + iOffset * 2;
	bool* gbBadShifts = m_gbBadShifts + iOffset;
	//---------------------------
	mGCorrect3D<<<m_aGridDim, m_aBlockDim, 0, m_aStreams[0]>>>(fBFactor,
	   gfUpsampled, m_gfPatCenters, gfPatShifts, 
	   gbBadShifts, m_iUpsample, gfPadAln);
	//---------------------------
	bool bNorm = true;
	m_pForwardFFT->Forward(gfPadAln, bNorm, m_aStreams[0]);
}

void GCorrectPatchShift::mMotionDecon(hipfftComplex* gCmpFrm)
{
}
